
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <png.h>
#include <string.h>
#include <stdlib.h>
#define MAX_ITERATION 100;

/* Calc Mandelbrot Set */
__global__ void kernel(int width, int height, float min_real, float min_imag, float max_real, float max_imag, int iteration, float *buffer);

int writeImage(const char* filename, int width, int height, float *buffer, const char* title);

static void setRGB(png_byte *ptr, float val);

/* ./exec <min_real> <min_imag> <max_real> <max_imag> <W> <H> <CPU/GPU> <Treadhs> <Saida> */
int main(int argc, char *argv[]){

	if(argc != 10){
		printf("Please specify output file\n");
		return 1;
	}

	/* Set variables */
	float min_real = atof(argv[1]);
	float min_imag = atof(argv[2]);
	float max_real = atof(argv[3]);
	float max_imag = atof(argv[4]);
	int width = atoi(argv[5]);
	int height = atoi(argv[6]);
	char accelerator[50];
	strcpy(accelerator, argv[7]);
	int threads = atoi(argv[8]);
	char file_name[50];
	strcpy(file_name, argv[9]);
	int iteration = MAX_ITERATION;
	float *buffer;
	float *d_buffer;


	/* Aloco no PC */
	buffer = (float*)malloc(width*height*sizeof(float));
	/* Aloco na placa de video */
	
	//printf("Alocando effer na placa de video\n");
	hipMalloc((void**)&d_buffer, width*height*sizeof(float));
	/* Não achei um cara pra verificar esta alocação */
	
	/* Setar os bocos para trabalhar na placa de video */
	dim3 block_size(16, 16);
	dim3 gred_size(width / block_size.x, height / block_size.y);
		
	//printf("Mandando calculo pra placa e video\n");
    	/* Se eu chamar a funcao em cuda */
	kernel<<<gred_size, block_size>>>(width, height, min_real, min_imag, max_imag, max_real, iteration, d_buffer);
	/* Passei N blocos, sendo uma thred por bloco, e por copia, nao parametro */
	
	hipDeviceSynchronize();

	/* Pego o buffer da Placa de video e trago para o PC */
	hipMemcpy(buffer, d_buffer, width*height*sizeof(float), hipMemcpyDeviceToHost);
	//printf("Trouxe da placa de vieo\n");	
	
	
	/* SEQUENCIAL ->
	float *buffer = mandelbrot_image(width, height, min_real, min_imag, max_real, max_imag, 100);
	if (buffer == NULL) {
		return 1;
	}*/



	printf("Saving PNG\n");
	int result = writeImage(file_name, width, height, buffer, "MANDELBROT");
	

	/* Free no buffer na placa de video */
	hipFree(buffer);
	/* Free no buffer no PC */
	free(buffer);

	return 0;
}

__global__ void kernel(int width, int height, float min_real, float min_imag, float max_real, float max_imag, int iteration_max, float *buffer){

	/* Isso acontece na main com copia para paca de video */
	/*float *buffer = (float*)malloc(width * height * sizeof(float));
	  if (buffer == NULL) {
	  printf("Could not create image buffer\n");
	  return NULL;
	  }*/
	int col = (blockIdx.y * blockDim.y) + threadIdx.y;
	int row = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idx = row * width + col;
	int id_i = (row*width+col)/width;
	int id_j = (col*width+col)/width;
	if(col >= width || row >= height) return;

	float x0 = ((float)row / width) * (max_real - min_real);
	float y0 = ((float)col / height) * (max_imag - min_imag);


	//int x_position, y_position;
	float minMu = iteration_max;
	float maxMu = 0;
	//float x0= (max_real - min_real) / width;
	//float y0= (max_imag - min_imag) / height;
	float z_x = 0.0f, z_y = 0.0f;
	//float z_x2, z_y2;
	int iteration = 0;
	float xtemp;

	while((z_x * z_x + z_y * z_y <= 4.0f) && iteration < iteration_max){
		xtemp = z_x * z_x - z_y * z_y + x0;
		z_y = 2.0f * z_x * z_y + y0;
		z_x = xtemp;
		iteration++;
	}
	
	int color = iteration  * 3;
	if(color >= 256) color =0;
	buffer[idx] = color;
	/*if (iteration < iteration_max) {
		float modZ = sqrt(z_x*z_x + z_y*z_y);
		float mu = iteration - (log(log(modZ))) / log(2);
		if (mu > maxMu)
			maxMu = mu;
		if (mu < minMu)
			minMu = mu;
		buffer[idx] = mu;
	}
	else {
		buffer[idx] = 0;
	}*/
	/*for (y_position = 0 ; y_position < height ; y_position++){
		//float y_pointer = max_imag - delt_y * y_position;
		for (x_position = 0 ; x_position < width ; x_position++){
			float y_pointer = max_imag - delt_y * y_position;
			float x_pointer = min_real + delt_x * x_position;

			z_x = x_pointer;
			z_y = y_pointer;
			z_x2 = z_x * z_x;
			z_y2 = z_y * z_y;

			for (iteration = 0; iteration < iteration_max && ((z_x2+z_y2) < 4); iteration++){
				z_y = 2 * z_x * z_y + y_pointer;
				z_x = z_x2 - z_y2 + x_pointer;
				z_x2 = z_x * z_x;
				z_y2 = z_y * z_y;
			}

			if (iteration < iteration_max) {
				float modZ = sqrt(z_x*z_x + z_y*z_y);
				float mu = iteration - (log(log(modZ))) / log(2);
				if (mu > maxMu)
					maxMu = mu;
				if (mu < minMu)
					minMu = mu;
				buffer[y_position * width + x_position] = mu;
			}
			else {
				buffer[y_position * width + x_position] = 0;
			}
		}
	}*/

	// Scale buffer values between 0 and 1
	/*int count = width * height;
	while (count) {
		count --;
		buffer[count] = (buffer[count] - minMu) / (maxMu - minMu);
	}*/

	/* Acho que função __global__ não tem retorno, tem que ser copiado da memoria da placa */
	//return buffer;
};


__host__ static void setRGB(png_byte *ptr, float val){
	int v = (int)(val * 767);
	if (v < 0) v = 0;
	if (v > 767) v = 767;
	int offset = v % 256;

	if (v<256) {
		ptr[0] = 0; ptr[1] = 0; ptr[2] = offset;
	}
	else if (v<512) {
		ptr[0] = 0; ptr[1] = offset; ptr[2] = 255-offset;
	}
	else {
		ptr[0] = offset; ptr[1] = 255-offset; ptr[2] = 0;
	}
};


__host__ int writeImage(const char* filename, int width, int height, float *buffer, const char* title){
	int code = 0;
	FILE *fp = NULL;
	png_structp png_ptr = NULL;
	png_infop info_ptr = NULL;
	png_bytep row = NULL;

	// Open file for writing (binary mode)
	fp = fopen(filename, "wb");
	if (fp == NULL) {
		printf( "Could not open file %s for writing\n", filename);
		code = 1;
		goto finalise;
	}

	// Initialize write structure
	png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
	if (png_ptr == NULL) {
		printf( "Could not allocate write struct\n");
		code = 1;
		goto finalise;
	}

	// Initialize info structure
	info_ptr = png_create_info_struct(png_ptr);
	if (info_ptr == NULL) {
		printf( "Could not allocate info struct\n");
		code = 1;
		goto finalise;
	}

	// Setup Exception handling
	if (setjmp(png_jmpbuf(png_ptr))) {
		printf( "Error during png creation\n");
		code = 1;
		goto finalise;
	}

	png_init_io(png_ptr, fp);

	// Write header (8 bit colour depth)
	png_set_IHDR(png_ptr, info_ptr, width, height,
			8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
			PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);


	png_write_info(png_ptr, info_ptr);

	// Allocate memory for one row (3 bytes per pixel - RGB)
	row = (png_bytep) malloc(3 * width * sizeof(png_byte));

	// Write image data
	int x, y;
	for (y=0 ; y<height ; y++) {
		for (x=0 ; x<width ; x++) {
			setRGB(&(row[x*3]), buffer[y*width + x]);
		}
		png_write_row(png_ptr, row);
	}

	// End write
	png_write_end(png_ptr, NULL);

	finalise:
	if (fp != NULL) fclose(fp);
	if (info_ptr != NULL) png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
	if (png_ptr != NULL) png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
	if (row != NULL) free(row);

	return code;
};
