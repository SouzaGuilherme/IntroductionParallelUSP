
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <png.h>
#include <string.h>
#include <stdlib.h>
#define MAX_ITERATION 100
#define CEIL(a, b) (((a) + (b) - 1)/(b)) 

__global__ void kernel(int width, int height, float min_real, float min_imag, float max_real, float max_imag, int iteration, float *buffer);

int writeImage(const char* filename, int width, int height, float *buffer, const char* title);

static void setRGB(png_byte *ptr, float val);

#define CEIL(a, b) (((a) + (b) - 1)/(b)) 

/* ./exec <min_real> <min_imag> <max_real> <max_imag> <W> <H> <CPU/GPU> <Treadhs> <Saida> */
int main(int argc, char *argv[]){

	if(argc != 10){
		printf("Please specify output file\n");
		return 1;
	}

	/* Set variables */
	float min_real = atof(argv[1]);
	float min_imag = atof(argv[2]);
	float max_real = atof(argv[3]);
	float max_imag = atof(argv[4]);
	int width = atoi(argv[5]);
	int height = atoi(argv[6]);
	char accelerator[50];
	strcpy(accelerator, argv[7]);
	int threads = atoi(argv[8]);
	char file_name[50];
	strcpy(file_name, argv[9]);
	int iteration = MAX_ITERATION;
	float *buffer;
	float *d_buffer;


	/* Aloco no PC */
	buffer = (float*)malloc(width*height*sizeof(float));
	
	/* Aloco na placa de video */
	hipMalloc(&d_buffer, width*height*sizeof(float));
	
	/* Setar os bocos para trabalhar na placa de video */
	int threads_per_block = threads;
	int num_blocks = CEIL((width*height), threads_per_block);
		
    	/* Se eu chamar a funcao em cuda */
	kernel<<<num_blocks, threads_per_block>>>(width, height, min_real, min_imag, max_imag, max_real, iteration, d_buffer);
	/* Passei N blocos, sendo uma x threads por bloco */
	
	hipDeviceSynchronize();

	/* Pego o buffer da Placa de video e trago para o PC */
	hipMemcpy(buffer, d_buffer, width*height*sizeof(float), hipMemcpyDeviceToHost);
	
	/* So printa a imagem */
	printf("Saving PNG\n");
	int result = writeImage(file_name, width, height, buffer, "MANDELBROT");
	

	/* Free no buffer na placa de video */
	hipFree(buffer);
	/* Free no buffer no PC */
	free(buffer);

	return 0;
}

__global__ void kernel(int width, int height, float min_real, float min_imag, float max_real, float max_imag, int iteration_max, float *buffer){

	int index_vector = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = index_vector / width;
	int j = index_vector % width;
	float del_x = (max_real - min_real)/width;
	float del_y= (max_imag - min_imag)/height;
	float z_y = 0.0;
	float z_x = 0.0; 
	int iteration = 0;
	float xtemp;
	float x_point = min_real + del_x * j;
	float y_point = max_imag - del_y * i;
	
	while((((z_x * z_x) + (z_y * z_y)) <= 4) && (iteration < iteration_max)){
		xtemp = ((z_x * z_x) - (z_y * z_y)+x_point);
		z_y = (2.0 * z_x * z_y)+ y_point;
		z_x = xtemp;
		iteration++;
	}
	
	buffer[index_vector] = iteration;
};

__host__ static void setRGB(png_byte *ptr, float val){

	int v = 255 - (int)(val/MAX_ITERATION) * 255;
	if(v == 0){
		ptr[0] = v;
		ptr[1] = v;
		ptr[2] = v;
	}else{
		if(val < 10){
			ptr[0] = 192;
			ptr[1] = 217;
			ptr[2] = 217;
		}else if(val < 15){
			ptr[0] = 95;
			ptr[1] = 159;
			ptr[2] = 159;
		}else if(val < 25){
			ptr[0] = 0;
			ptr[1] = 255;
			ptr[2] = 255;
		}else if(val < 50){
			ptr[0] = 255;
			ptr[1] = 0;
			ptr[2] = 255;
		}else if(val < 75){
			ptr[0] = 234;
			ptr[1] = 173;
			ptr[2] = 234;
		}else{
			ptr[0] = 79;
			ptr[1] = 47;
			ptr[2] = 79;
		}
	}
}; 


__host__ int writeImage(const char* filename, int width, int height, float *buffer, const char* title){
	int code = 0;
	FILE *fp = NULL;
	png_structp png_ptr = NULL;
	png_infop info_ptr = NULL;
	png_bytep row = NULL;

	// Open file for writing (binary mode)
	fp = fopen(filename, "wb");
	if (fp == NULL) {
		printf( "Could not open file %s for writing\n", filename);
		code = 1;
		goto finalise;
	}

	// Initialize write structure
	png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
	if (png_ptr == NULL) {
		printf( "Could not allocate write struct\n");
		code = 1;
		goto finalise;
	}

	// Initialize info structure
	info_ptr = png_create_info_struct(png_ptr);
	if (info_ptr == NULL) {
		printf( "Could not allocate info struct\n");
		code = 1;
		goto finalise;
	}

	// Setup Exception handling
	if (setjmp(png_jmpbuf(png_ptr))) {
		printf( "Error during png creation\n");
		code = 1;
		goto finalise;
	}

	png_init_io(png_ptr, fp);

	// Write header (8 bit colour depth)
	png_set_IHDR(png_ptr, info_ptr, width, height,
			8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
			PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);


	png_write_info(png_ptr, info_ptr);

	// Allocate memory for one row (3 bytes per pixel - RGB)
	row = (png_bytep) malloc(3 * width * sizeof(png_byte));

	// Write image data
	int x, y;
	for (y=0 ; y<height ; y++) {
		for (x=0 ; x<width ; x++) {
			setRGB(&(row[x*3]), buffer[y*width + x]);
		}
		png_write_row(png_ptr, row);
	}

	// End write
	png_write_end(png_ptr, NULL);

	finalise:
	if (fp != NULL) fclose(fp);
	if (info_ptr != NULL) png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
	if (png_ptr != NULL) png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
	if (row != NULL) free(row);

	return code;
};
