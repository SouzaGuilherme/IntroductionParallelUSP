
#include <hip/hip_runtime.h>
#include<stdio.h>
#define iteration_max 100
#define CEIL(a, b) (((a) + (b) - 1)/(b))

// Function using of the internet
////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
	if( hipSuccess != err) {
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
				file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);

	}
}
/* Traz a GLOBAL para cá */
/* Calc_mandelbrot NVIDIA_CUDA */
__global__ void kernel(int width, int height, float min_real, float min_imag, float max_real, float max_imag, int *buffer){
	int index_vector = (blockIdx.x * blockDim.x) + threadIdx.x;
	int val_for_cal = buffer[index_vector];
	int i = val_for_cal / width;
	int j = val_for_cal % width;
	float del_x = (max_real - min_real)/width;
	float del_y= (max_imag - min_imag)/height;
	int iteration = 0;
	float x_point = min_real + del_x * i;
	float y_point = max_imag - del_y * j;
	float z_y = y_point;
	float z_x = x_point;
	float z_x2 = z_x * z_x;
	float z_y2 = z_y * z_y;

	for(iteration = 0; iteration < iteration_max && ((z_x2 + z_y2) <= 4); iteration++){
		z_y = (2.0 * z_x * z_y)+ y_point;
		z_x = z_x2 - z_y2 + x_point;
		z_x2 = z_x * z_x;
		z_y2 = z_y * z_y;
	}
	buffer[index_vector] = iteration;
};


extern "C" void invoke_cuda(int tam_vetor, int width, int height, float *min_real, float *min_imag, float *max_real, float *max_imag, int threads, int *vetor){
	int *d_vetor;
	
	/* Aloco na placa de video */
	checkCudaErrors(hipMalloc(&d_vetor, tam_vetor*sizeof(int)));


	checkCudaErrors(hipMemcpy(d_vetor, vetor, tam_vetor*sizeof(int), hipMemcpyHostToDevice));
	
	/* Setar os bocos para trabalhar na placa de video */
	int threads_per_block = threads;
	int num_blocks = CEIL((tam_vetor), threads_per_block);
		
	kernel<<<num_blocks, threads_per_block>>>(width, height, *min_real, *min_imag, *max_real, *max_imag, d_vetor);

	checkCudaErrors(hipDeviceSynchronize());
        
	/* Pego o buffer da Placa de video e trago para o PC */
	checkCudaErrors(hipMemcpy(vetor, d_vetor, tam_vetor*sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipFree(d_vetor));
}
